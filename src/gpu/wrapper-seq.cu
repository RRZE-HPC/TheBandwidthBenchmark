#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "kernels.h"
#include "util.h"
#include "gpu.h" 
#include "timing.h" 

#define HARNESS(kernel)                                                        \
  double time = 0.0;                                                           \
  for( int i = 0 ; i < numDevices ; ++i )                                      \
  {                                                                            \
    GPU_ERROR(hipSetDevice(i));                                               \
    GPU_ERROR(hipFree(0));                                                    \
    double S = getTimeStamp();                                                 \
    for (int j = 0; j < iter; j++) {                                           \
      kernel;                                                                  \
    }                                                                          \
    GPU_ERROR(hipDeviceSynchronize());                                        \
    double E = getTimeStamp();                                                 \
    time = E - S;                                                              \
  }                                                                            \
  return (time/numDevices);



extern "C" double init_seq_wrapper(double *__restrict__ b[], int scalar, size_t N, int iter) {

  HARNESS((init<<<N / thread_block_size + 1, thread_block_size>>>(b[i], scalar, N)))

}

extern "C" double copy_seq_wrapper(double *__restrict__ c[], double *__restrict__ a[], size_t N, int iter) {

  HARNESS((copy<<<N / thread_block_size + 1, thread_block_size>>>(c[i], a[i], N)))

}

extern "C" double sum_seq_wrapper(double *__restrict__ a[], size_t N, int iter) {

  // HARNESS((sum<<<N / thread_block_size + 1, thread_block_size>>>(a[i], N)))

}

extern "C" double update_seq_wrapper(double *__restrict__ a[], int scalar, size_t N, int iter) {

  HARNESS((update<<<N / thread_block_size + 1, thread_block_size>>>(a[i], scalar, N)))

}

extern "C" double triad_seq_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], int scalar, size_t N, int iter) {

  HARNESS((triad<<<N / thread_block_size + 1, thread_block_size>>>(a[i], b[i], c[i], scalar, N)))

}

extern "C" double daxpy_seq_wrapper(double *__restrict__ a[], double *__restrict__ b[], int scalar, size_t N, int iter) {

  HARNESS((daxpy<<<N / thread_block_size + 1, thread_block_size>>>(a[i], b[i], scalar, N)))

}

extern "C" double striad_seq_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N, int iter) {

  HARNESS((striad<<<N / thread_block_size + 1, thread_block_size>>>(a[i], b[i], c[i], d[i], N)))

}

extern "C" double sdaxpy_seq_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], size_t N, int iter) {

  HARNESS((sdaxpy<<<N / thread_block_size + 1, thread_block_size>>>(a[i], b[i], c[i], N)))

}