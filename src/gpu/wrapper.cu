#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "kernels.h"
#include "util.h"
#include "gpu.h" 
#include "timing.h" 

#ifdef _OPENMP
#define OMP_PARALLEL _Pragma("omp parallel for num_threads(numDevices) reduction(max:time)")
#else
#define OMP_PARALLEL
#endif

#define HARNESS(kernel)                                                        \
  OMP_PARALLEL                                                                 \
  for( int i = 0 ; i < numDevices ; ++i )                                      \
  {                                                                            \
    GPU_ERROR(hipSetDevice(i));                                               \
    GPU_ERROR(hipFree(0));                                                    \
    double S = getTimeStamp();                                                 \
    kernel;                                                                    \
    GPU_ERROR(hipDeviceSynchronize());                                        \
    double E = getTimeStamp();                                                 \
    time = E - S;                                                              \
  }                                                                         

extern "C" void initArrays(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N) {

#ifdef _OPENMP
  #pragma omp parallel for num_threads(numDevices)
#endif
  for( int i = 0 ; i < numDevices ; ++i )
  {    
    GPU_ERROR(hipSetDevice(i));
    GPU_ERROR(hipFree(0));

    init_all<<<N / 1024 + 1, 1024>>>(a[i], b[i], c[i], d[i], N);

    GPU_ERROR(hipDeviceSynchronize());
  }

}

extern "C" double init_wrapper(double *__restrict__ b[], int scalar, size_t N) {

  double time = 0.0;

  HARNESS((init<<<N / 1024 + 1, 1024>>>(b[i], scalar, N)))

  return time;
}

extern "C" double copy_wrapper(double *__restrict__ c[], double *__restrict__ a[], size_t N) {

  double time = 0.0;

  HARNESS((copy<<<N / 1024 + 1, 1024>>>(c[i], a[i], N)))

  return time;
}

extern "C" double update_wrapper(double *__restrict__ a[], int scalar, size_t N) {

  double time = 0.0;

  HARNESS((update<<<N / 1024 + 1, 1024>>>(a[i], scalar, N)))

  return time;
}


extern "C" double triad_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], int scalar, size_t N) {

  double time = 0.0;

  HARNESS((triad<<<N / 1024 + 1, 1024>>>(a[i], b[i], c[i], scalar, N)))

  return time;
}

extern "C" double daxpy_wrapper(double *__restrict__ a[], double *__restrict__ b[], int scalar, size_t N) {

  double time = 0.0;

  HARNESS((daxpy<<<N / 1024 + 1, 1024>>>(a[i], b[i], scalar, N)))

  return time;
}

extern "C" double striad_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N) {

  double time = 0.0;

  HARNESS((striad<<<N / 1024 + 1, 1024>>>(a[i], b[i], c[i], d[i], N)))

  return time;
}

extern "C" double sdaxpy_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], size_t N) {

  double time = 0.0;

  HARNESS((sdaxpy<<<N / 1024 + 1, 1024>>>(a[i], b[i], c[i], N)))

  return time;
}