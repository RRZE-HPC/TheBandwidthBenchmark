#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "kernels.h"
#include "wrapper.h"
#include "util.h"
#include "gpu.h" 
#include "timing.h" 


#ifdef _OPENMP
#define OMP_PARALLEL _Pragma("omp parallel for num_threads(numDevices) reduction(+:time)")
#else
#define OMP_PARALLEL
#endif

#define SHARED_MEM(kernel_name) getSharedMemSize(thread_block_size, thread_blocks_per_streaming_multiprocessor, (const void*)&kernel_name) 

#define HARNESS(kernel, kernel_name)                                           \
  double time = 0.0;                                                           \
  int shared_mem_size = SHARED_MEM(kernel_name);                               \
  OMP_PARALLEL                                                                 \
  for( int i = 0 ; i < numDevices ; ++i )                                      \
  {                                                                            \
    GPU_ERROR(hipSetDevice(i));                                               \
    GPU_ERROR(hipFree(0));                                                    \
    double S = getTimeStamp();                                                 \
    kernel;                                                                    \
    GPU_ERROR(hipDeviceSynchronize());                                        \
    double E = getTimeStamp();                                                 \
    time = E - S;                                                              \
  }                                                                            \
  return (time/numDevices);


extern "C" void initArrays(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N) {

#ifdef _OPENMP
  #pragma omp parallel for num_threads(numDevices)
#endif
  for( int i = 0 ; i < numDevices ; ++i )
  {    
    GPU_ERROR(hipSetDevice(i));
    GPU_ERROR(hipFree(0));

    init_all<<<N / thread_block_size + 1, thread_block_size>>>(a[i], b[i], c[i], d[i], N);

    GPU_ERROR(hipDeviceSynchronize());
  }

}

extern "C" double init_wrapper(double *__restrict__ b[], int scalar, size_t N) {

  HARNESS((init<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(b[i], scalar, N)), init)

}

extern "C" double copy_wrapper(double *__restrict__ c[], double *__restrict__ a[], size_t N) {

  HARNESS((copy<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(c[i], a[i], N)), copy)

}

extern "C" double update_wrapper(double *__restrict__ a[], int scalar, size_t N) {

  HARNESS((update<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(a[i], scalar, N)), update)

}

extern "C" double triad_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], int scalar, size_t N) {

  HARNESS((triad<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(a[i], b[i], c[i], scalar, N)), triad)

}

extern "C" double daxpy_wrapper(double *__restrict__ a[], double *__restrict__ b[], int scalar, size_t N) {

  HARNESS((daxpy<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(a[i], b[i], scalar, N)), daxpy)

}

extern "C" double striad_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N) {

  HARNESS((striad<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(a[i], b[i], c[i], d[i], N)), striad)

}

extern "C" double sdaxpy_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], size_t N) {

  HARNESS((sdaxpy<<<N / thread_block_size + 1, thread_block_size, shared_mem_size>>>(a[i], b[i], c[i], N)), sdaxpy)

}

extern "C" double sum_wrapper(double *__restrict__ a[], size_t N) {

  double time = 0.0;
  OMP_PARALLEL
  for( int i = 0 ; i < numDevices ; ++i )
  {    
    GPU_ERROR(hipSetDevice(i));
    GPU_ERROR(hipFree(0));
    double *a_out;
    GPU_ERROR(hipMalloc(&a_out, (N  + (thread_block_size - 1)) / thread_block_size * sizeof(double)));
    double S = getTimeStamp();
    sum<<<N / (thread_block_size * 2) + 1, thread_block_size, thread_block_size * sizeof(double)>>>(a[i], a_out, N);
    GPU_ERROR(hipDeviceSynchronize());
    double E = getTimeStamp();
    time = E - S;
  }
  return (time/numDevices);

}

extern "C" void setBlockSize()
{
  hipDeviceProp_t prop;
  GPU_ERROR(hipGetDeviceProperties(&prop, 0));

  max_thread_block_size = prop.maxThreadsPerBlock;
  max_threads_per_streaming_multiprocessor = prop.maxThreadsPerMultiProcessor;

  // Not the best case for thread_block_size. 
  // Varying thread_block_size can result in 
  // better performance and thread occupancy.
  thread_block_size = max_thread_block_size;

#ifdef THREADBLOCKSIZE
  thread_block_size = THREADBLOCKSIZE;
#endif

  thread_blocks_per_streaming_multiprocessor = floor(max_threads_per_streaming_multiprocessor/thread_block_size);

#ifdef THREADBLOCKPERSM
  thread_blocks_per_streaming_multiprocessor = MIN(thread_blocks_per_streaming_multiprocessor, THREADBLOCKPERSM);
#endif

  occupancy = (((double)thread_block_size * (double)thread_blocks_per_streaming_multiprocessor)/ (double)max_threads_per_streaming_multiprocessor)*100;
}

int getSharedMemSize(int thread_block_size, int thread_blocks_per_sm, const void* func) {

#ifdef THREADBLOCKPERSM
  int max_active_thread_blocks = 0;
  int shared_mem_size = 1024;

  GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &max_active_thread_blocks, func, thread_block_size, shared_mem_size));

  while (max_active_thread_blocks > thread_blocks_per_sm) {
    shared_mem_size += 256;
    GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_thread_blocks, func, thread_block_size, shared_mem_size));
  }
  return shared_mem_size;
#else
  return 1;
#endif

}
