#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "kernels.h"
#include "util.h"


__global__ void init_all(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, double *__restrict__ d, const size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] = 2.0;
  b[tidx] = 2.0;
  c[tidx] = 0.5;
  d[tidx] = 1.0;

}

__global__ void init(double *__restrict__ b, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  b[tidx] = scalar;

}

__global__ void copy(double *__restrict__ c, double *__restrict__ a, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  c[tidx] = a[tidx];

}

__global__ void update(double *__restrict__ a, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] = a[tidx] * scalar;

}

__global__ void triad(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  b[tidx] + scalar * c[tidx];

}

__global__ void daxpy(double *__restrict__ a, double *__restrict__ b, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  a[tidx] + scalar * b[tidx];

}

__global__ void striad(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, double *__restrict__ d, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  b[tidx] + d[tidx] * c[tidx];

}

__global__ void sdaxpy(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  a[tidx] + b[tidx] * c[tidx];

}