#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "kernels.h"
#include "util.h"

int thread_block_size = 1;
int max_thread_block_size = 1;
int max_threads_per_streaming_multiprocessor = 1;
int thread_blocks_per_streaming_multiprocessor = 1;
double occupancy = 0.0;

__global__ void init_all(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, double *__restrict__ d, const size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] = 2.0;
  b[tidx] = 2.0;
  c[tidx] = 0.5;
  d[tidx] = 1.0;

}

__global__ void init(double *__restrict__ b, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  b[tidx] = scalar;

}

__global__ void copy(double *__restrict__ c, double *__restrict__ a, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  c[tidx] = a[tidx];

}

__global__ void update(double *__restrict__ a, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] = a[tidx] * scalar;

}

__global__ void triad(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  b[tidx] + scalar * c[tidx];

}

__global__ void daxpy(double *__restrict__ a, double *__restrict__ b, int scalar, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  a[tidx] + scalar * b[tidx];

}

__global__ void striad(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, double *__restrict__ d, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  b[tidx] + d[tidx] * c[tidx];

}

__global__ void sdaxpy(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, size_t N) {
  
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (tidx >= N)
    return;

  a[tidx] =  a[tidx] + b[tidx] * c[tidx];

}

__device__ void warpReduce(volatile int* shared_data, int tidx){
  shared_data[tidx] += shared_data[tidx + 32];
  shared_data[tidx] += shared_data[tidx + 16];
  shared_data[tidx] += shared_data[tidx + 8];
  shared_data[tidx] += shared_data[tidx + 4];
  shared_data[tidx] += shared_data[tidx + 2];
  shared_data[tidx] += shared_data[tidx + 1];
}

__global__ void sum(double *__restrict__ a, double *__restrict__ a_out, size_t N){
    extern __shared__ int shared_data[];

    unsigned int tidx = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    shared_data[tidx] = a[i] + a[i + blockDim.x];
    __syncthreads();

    for(int s = blockDim.x/2; s > 32; s >>= 1) { 

        if (tidx < s){  
            shared_data[tidx] += shared_data[tidx + s];
        }
        __syncthreads();
    }

    if (tidx < 32){
      warpReduce(shared_data, tidx);
    }

    if (tidx == 0){
        a[blockIdx.x] = shared_data[0];
    }
}