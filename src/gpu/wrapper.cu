#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "kernels.h"
#include "util.h"
#include "gpu.h" 
#include "timing.h" 

#ifdef _OPENMP
#define OMP_PARALLEL _Pragma("omp parallel for num_threads(numDevices) reduction(+:time)")
#else
#define OMP_PARALLEL
#endif

int threadBlockSize = 1;
int maxThreadBlockSize = 1;
int maxThreadsPerStreamingMultiprocessor = 1;
int threadBlocksPerStreamingMultiprocessor = 1;
double occupancy = 0.0;

#define HARNESS(kernel)                                                        \
  double time = 0.0;                                                           \
  OMP_PARALLEL                                                                 \
  for( int i = 0 ; i < numDevices ; ++i )                                      \
  {                                                                            \
    GPU_ERROR(hipSetDevice(i));                                               \
    GPU_ERROR(hipFree(0));                                                    \
    double S = getTimeStamp();                                                 \
    kernel;                                                                    \
    GPU_ERROR(hipDeviceSynchronize());                                        \
    double E = getTimeStamp();                                                 \
    time = E - S;                                                              \
  }                                                                            \
  return (time/numDevices);


extern "C" void initArrays(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N) {

#ifdef _OPENMP
  #pragma omp parallel for num_threads(numDevices)
#endif
  for( int i = 0 ; i < numDevices ; ++i )
  {    
    GPU_ERROR(hipSetDevice(i));
    GPU_ERROR(hipFree(0));

    init_all<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], b[i], c[i], d[i], N);

    GPU_ERROR(hipDeviceSynchronize());
  }

}

extern "C" double init_wrapper(double *__restrict__ b[], int scalar, size_t N) {

  HARNESS((init<<<N / threadBlockSize + 1, threadBlockSize>>>(b[i], scalar, N)))

}

extern "C" double copy_wrapper(double *__restrict__ c[], double *__restrict__ a[], size_t N) {

  HARNESS((copy<<<N / threadBlockSize + 1, threadBlockSize>>>(c[i], a[i], N)))

}

extern "C" double sum_wrapper(double *__restrict__ c[], double *__restrict__ a[], size_t N) {

  HARNESS((sum<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], N)))

}

extern "C" double update_wrapper(double *__restrict__ a[], int scalar, size_t N) {

  HARNESS((update<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], scalar, N)))

}

extern "C" double triad_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], int scalar, size_t N) {

  HARNESS((triad<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], b[i], c[i], scalar, N)))

}

extern "C" double daxpy_wrapper(double *__restrict__ a[], double *__restrict__ b[], int scalar, size_t N) {

  HARNESS((daxpy<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], b[i], scalar, N)))

}

extern "C" double striad_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], double *__restrict__ d[], size_t N) {

  HARNESS((striad<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], b[i], c[i], d[i], N)))

}

extern "C" double sdaxpy_wrapper(double *__restrict__ a[], double *__restrict__ b[], double *__restrict__ c[], size_t N) {

  HARNESS((sdaxpy<<<N / threadBlockSize + 1, threadBlockSize>>>(a[i], b[i], c[i], N)))

}

extern "C" void setBlockSize()
{
  hipDeviceProp_t prop;
  GPU_ERROR(hipGetDeviceProperties(&prop, 0));

  maxThreadBlockSize = prop.maxThreadsPerBlock;
  maxThreadsPerStreamingMultiprocessor = prop.maxThreadsPerMultiProcessor;
  int maxBlocksPerSM = prop.maxBlocksPerMultiProcessor;
  int warpSize = prop.warpSize; 
  printf("TB: %d\nWP: %d, TperSM: %d\nBperSM: %d\n",maxThreadBlockSize, warpSize, maxThreadsPerStreamingMultiprocessor, maxBlocksPerSM);

  // Assuming that maxThreadsPerMultiProcessor is multiple of 32 and even.
  // For max occupancy, threadBlockSize is divided into 2 so that GPU can 
  // schedule 2 thread block per Streaming Multiprocessor.
  threadBlockSize = maxThreadBlockSize;

#ifdef BLOCKSIZE
  threadBlockSize = BLOCKSIZE;
#endif

  threadBlocksPerStreamingMultiprocessor = floor(maxThreadsPerStreamingMultiprocessor/threadBlockSize);
  occupancy = (((double)threadBlockSize * (double)threadBlocksPerStreamingMultiprocessor)/ (double)maxThreadsPerStreamingMultiprocessor)*100;
}